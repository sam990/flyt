#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)


#define CHUNK_SIZE (1024llu * 1024llu * 4llu)
#define TOTAL_MEM_ALLOC_MB 512llu
#define TOTAL_MEM_ALLOC ((TOTAL_MEM_ALLOC_MB) * 1024llu * 1024llu)
#define NUM_CHUNKS ((TOTAL_MEM_ALLOC) / CHUNK_SIZE)

int main() {
    
    void** chunk_list = (void **)malloc(sizeof(void*) * NUM_CHUNKS);

    for (size_t i = 0; i < NUM_CHUNKS; i++) {
        HANDLE_ERROR(hipMalloc(&chunk_list[i], CHUNK_SIZE));
    }

    for (size_t i = 0; i < NUM_CHUNKS; i++) {
        HANDLE_ERROR(hipFree(chunk_list[i]));
    }

    free(chunk_list);

    return 0;
}
